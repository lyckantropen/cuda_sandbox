#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <valarray>

constexpr size_t N_BLOCKS = 64;
constexpr size_t N_THREADS = 256;

__global__ void add_vec(double* a, double* b, double* res) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < N_BLOCKS * N_THREADS) {
        res[idx] = a[idx] + b[idx];
    }
}

int main() {
    constexpr size_t N = N_BLOCKS * N_THREADS;
    auto hst_a = std::vector<double>(N, 1.0);
    auto hst_b = std::vector<double>(N, 2.0);
    auto hst_res = std::vector<double>(N, 0.0);

    double *dev_a, *dev_b, *dev_res;
    hipMalloc(&dev_a, N * sizeof(double));
    hipMalloc(&dev_b, N * sizeof(double));
    hipMalloc(&dev_res, N * sizeof(double));

    hipMemcpy(dev_a, hst_a.data(), N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, hst_b.data(), N * sizeof(double), hipMemcpyHostToDevice);

    add_vec<<<N_BLOCKS, N_THREADS>>>(dev_a, dev_b, dev_res);
    hipDeviceSynchronize();

    hipMemcpy(hst_res.data(), dev_res, N * sizeof(double), hipMemcpyDeviceToHost);

    // validate
    auto val_res = std::valarray<double>(hst_res.data(), N);
    auto val_ab = std::valarray<double>(hst_a.data(), N) + std::valarray<double>(hst_b.data(), N);

    std::cout << std::ranges::equal(val_res, val_ab) << std::endl;

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_res);

    return 0;
}